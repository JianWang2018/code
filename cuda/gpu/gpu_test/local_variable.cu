#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void kernel() {
    double  a = 2.71828;    //register variables, automatic
    double  c[100];         //local variable, automatic
    __shared__ double b;    //shared variable
    int  tx  = threadIdx.x; //register variable
    if (tx == 0) {
        b = 3.1415926f;
    }
    //__syncthreads();        // run with/without this line
    printf("id = %d, a=%7.5f, b=%9.7f\n", tx, a, b);
}
int main() {
    kernel<<<1,8>>>();
    hipDeviceReset();
    return 0;
}
