#include <stdlib.h>

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <sys/time.h>



#define NUM_STEPS    2048

#define BLOCK_SIZE   128

#define MAX_OPTIONS  1024

#define ELEMS_PER_THREAD (NUM_STEPS/BLOCK_SIZE)

/* at T_N = NUM_STEPS, there are (NUM_STEPS + 1) leaves*/



double cpuSecond( ) {

    double sec;

    struct timeval tp;

    gettimeofday(&tp, NULL);

    sec = (double) tp.tv_sec + (double) tp.tv_usec*1e-6;

    return sec;

}



// cumulative standard normal distribution

__device__ __host__ double CND(double d) {



    const double b  =  0.2316419;

    const double a1 =  0.31938153;

    const double a2 = -0.356563782;

    const double a3 =  1.781477937;

    const double a4 = -1.821255978;

    const double a5 =  1.330274429;



    const double rsqrt2pi  = 0.3989422804014327;

    double K = 1.0/(1.0 + b*abs(d));



    double ans;



    ans  = K*(a1 + K*(a2 + K*(a3 + K*(a4 + K*a5) ) ) );

    ans *= rsqrt2pi*exp(-d*d/2.0);

    if (d > 0) {

        ans = 1.0 - ans;

    }

    return ans;

}



__host__ double

BlackScholes(double S0, double X, double r, double v, double T) {



    double sqrtT = sqrt(T);

    double d1    = ( log(S0/X) + (r + 0.5*v*v)*T )/(v*sqrtT);

    double d2    = d1 - v*sqrtT;

    double expRT = exp(-r*T);

    double CE;



    CE = S0*CND(d1) - X*expRT*CND(d2);

    return CE;

}





/* CPU verion of the binomial model */

__host__ double 

binomial(double S, double X, double r, double v, double T, int N, double* V ) {

// S current stock price; X strike price; 

// r, v the risk-free interest rate and volatility;

// T the expiry (unit yr); N the number of time steps in the binomial model.

// V[N+1] is the array to do the iteration;

// In the end, V[0] stores the price of the call option



    double  dt    = T/(double) N;

    double  vdt   = v*sqrt(dt);

    double  u     = exp( vdt );

    double  d     = 1.0/u;

    double  disc  = exp( r * dt );       // discounting factor

    double  discr = 1.0/disc;

    double  pu    = (disc - d)/(u - d);  // risk-neutral/martingale probability

    double  pd    = 1.0 - pu;

    int     i, j;

    double  Si;                          // intermediate stock price in the node



    // initialize the CALL option value on expiry;

    for (i = 0; i <= N; i++) {

        Si   = S * exp( vdt*(2*i - N) );     	// S[0] lowest stock price at expiry. 

        V[i] = (Si - X > 0) ? (Si - X) : 0;     // Call option, use "max(X-Si, 0.0)" for puts  

    }



    // iterate backward of the binomial tree (j the time step).

    for (j = N-1; j >= 0; j--) {

        for (i = 0; i <= j; i++) {

            V[i] =  (pd * V[i] + pu * V[i+1]) * discr;

        }

    }



    return V[0];

}



/* 1D grid, and 1D block; blocksize << NUM_STEPS

   each block  deals with only one option 

   each thread deals with part of the binomial tree */

__global__ void

binomialGPUv5(double* Sptr, double* Xptr, double* Cptr,

              double r, double v, double T) {



    int     tx    = threadIdx.x;

    int     bx    = blockIdx.x;

    double  S     = Sptr[bx];

    double  X     = Xptr[bx];

    double  dt    = T/(double) NUM_STEPS;

    double  vdt   = v*sqrt(dt);

    double  u     = exp(vdt);

    double  d     = 1.0/u;

    double  disc  = exp( r * dt );       // discounting factor

    double  discr = 1.0/disc;

    double  pu    = (disc - d)/(u - d);  // risk-neutral/martingale probability

    double  pd    = 1.0 - pu;

    

    int     i, j, k;

    double  Si;



    double  call_loc[ELEMS_PER_THREAD + 1]; //local array

    __shared__ 

    double  call_bound[BLOCK_SIZE + 1];



    for (i = 0; i < ELEMS_PER_THREAD; i++) {

        k  = tx * ELEMS_PER_THREAD + i;

        Si = S * exp( vdt * (2*k - NUM_STEPS) );

        call_loc[i] =  (Si - X > 0) ? (Si - X) : 0;

    }



    if (tx == BLOCK_SIZE - 1) {

        Si = S * exp( vdt * NUM_STEPS);

        call_bound[BLOCK_SIZE] = (Si - X > 0) ? (Si - X) : 0;

    }

    call_bound[tx] = call_loc[0];

    __syncthreads();

    call_loc[ELEMS_PER_THREAD] = call_bound[tx + 1];

    __syncthreads();



    for (j = 0; j < (BLOCK_SIZE - tx)*ELEMS_PER_THREAD; j++) {



        for (i = 0; i < ELEMS_PER_THREAD; i++) {

            call_loc[i] =  (pd*call_loc[i] + pu*call_loc[i+1])*discr;

        }

        call_bound[tx] = call_loc[0];

        __syncthreads();

        call_loc[ELEMS_PER_THREAD] = call_bound[tx + 1];

        __syncthreads();  // must be there, otherwise, fail

    }



    if (tx == 0) {

        Cptr[bx] = call_loc[0];

    }

}



/* general uniform random number between [low, high] */

double UniRand(double low, double high) {



    double t = (double) rand() / (double) RAND_MAX;

    return (1.0 - t)*low + t*high;

}



/* test the GPU binomial model for European Call Pricing */

int main(int argc, char** argv) {



    int      OPT_N   = MAX_OPTIONS;

    double * Sptr_h;

    double * Sptr_d;

    double * Xptr_h;

    double * Xptr_d;

    double * Cptr_h;

    double * Cptr_d;

    double * Cptr_c;

    double * Cptr_b;  		// from the Black-Scholes model

    int      size = OPT_N*sizeof(double);    

    int      i;

  

    double * Vptr_h; // used by the CPU verions





    double   r = 0.02;  // risk-free rate

    double   v = 0.30;  // volatility

    double   T = 1.00;  // maturity



    hipError_t error;

    double   tStart;

    double   tStop1, tStop2;



    double   sumDel = 0, sumRef = 0;



    Sptr_h = (double *) malloc(size); // input

    Xptr_h = (double *) malloc(size); // input

    Cptr_h = (double *) malloc(size); // CPU version output

    Cptr_c = (double *) malloc(size); // GPU version output

    Cptr_b = (double *) malloc(size); // CPU Black-Scholes output



    Vptr_h = (double *) malloc((NUM_STEPS+1)*sizeof(double));



    for (i = 0; i < OPT_N; i++) {

        Sptr_h[i] = UniRand(5.0, 30.0);

        Xptr_h[i] = UniRand(1.0, 40.0);

    }



    hipMalloc( (void **) &Sptr_d, size);

    hipMalloc( (void **) &Xptr_d, size);

    hipMalloc( (void **) &Cptr_d, size);



    hipMemcpy(Sptr_d, Sptr_h, size, hipMemcpyHostToDevice);

    hipMemcpy(Xptr_d, Xptr_h, size, hipMemcpyHostToDevice);



    dim3 dimGrid(MAX_OPTIONS);

    dim3 dimBlock(BLOCK_SIZE);

 

    printf("Depth of the tree NUM_STEPS = %d\n", NUM_STEPS); 

    printf("grid structure: <<<%d, %d>>>\n", dimGrid.x, dimBlock.x);

    printf("Starting the GPU code...\n");



    tStart = cpuSecond();

    hipDeviceSynchronize();

    binomialGPUv5<<<dimGrid, dimBlock>>>(Sptr_d, Xptr_d, Cptr_d, r, v, T);

    hipMemcpy(Cptr_c, Cptr_d, size, hipMemcpyDeviceToHost);   

    error = hipPeekAtLastError();

    if (error != hipSuccess) {

        printf("GPU code failed %s\n", hipGetErrorString(error) ); 

        exit(-1);

    } else {

        tStop1 = cpuSecond() - tStart;

        printf("GPU code finished within %12.6f seconds\n", tStop1);

    }



    tStart = cpuSecond();

    for (i = 0; i < OPT_N; i++) {

        Cptr_h[i] = binomial(Sptr_h[i], Xptr_h[i], r, v, T, NUM_STEPS, Vptr_h);

    }

    tStop2  = cpuSecond() - tStart;

    printf("CPU code finished within %12.6f seconds\n", tStop2);

    printf("Speed up you got %8.2f\n", tStop2/tStop1);



    for (i = 0; i < OPT_N; i++) {

        Cptr_b[i] = BlackScholes(Sptr_h[i], Xptr_h[i], r, v, T);

    }



    printf("Compare the GPU and CPU binary model now...\n");

    for (i = 0; i < OPT_N; i++) {

        if ( abs( Cptr_h[i] - Cptr_c[i] ) > 1e-8 ) {

            printf("%d %8.2f %8.2f %12.6f %12.6f \n", i, Sptr_h[i], Xptr_h[i], Cptr_h[i], Cptr_c[i]);      

            printf("comparion failed\n");

	    exit(-1);

        } 

    }

    printf("passed comparison between GPU and CPU binomial model\n");



    printf("Compare the binary model with Black-Scholes model now...\n");

    for (i = 0; i < OPT_N; i++) {

        sumDel += abs( Cptr_h[i] - Cptr_b[i] ); 

        sumRef += Cptr_h[i];

    }

    if ( sumDel/sumRef > 1e-4 ) {

        printf("Black-Scholes VS Binomial comparison failed\n");

	exit(-1);

    } 

    printf("passed comparison between binomial and Black-Scholes model\n");



    printf("Here are outputs for the first 10 lines\n");

    for (i = 0; i < 10; i++) {

        printf("%8.2f, %8.2f, %10.4f, %10.4f, %10.4f\n", 

              Sptr_h[i], Xptr_h[i], Cptr_c[i], Cptr_h[i], Cptr_b[i]);

    }



    free(Sptr_h); free(Xptr_h); free(Cptr_h); free(Cptr_c); free(Cptr_b); free(Vptr_h);

    hipFree(Sptr_d); hipFree(Xptr_d); hipFree(Cptr_d);

    hipDeviceReset();



    return(0);

}


