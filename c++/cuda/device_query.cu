#include <stdio.h>
#include <hip/hip_runtime.h>
int main( ) {
    int dev = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    printf("device id %d, name %s\n", dev, prop.name);
    printf("number of multi-processors = %d\n", 
        prop.multiProcessorCount);
    printf("Total constant memory: %4.2f kb\n", 
        prop.totalConstMem/1024.0);
    printf("Shared memory per block: %4.2f kb\n",
        prop.sharedMemPerBlock/1024.0);
    printf("Total registers per block: %d\n", 
        prop.regsPerBlock);
    printf("Maximum threads per block: %d\n", 
        prop.maxThreadsPerBlock);
    printf("Maximum threads per multi-processor: %d\n", 
        prop.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multi-processor %d\n",
        prop.maxThreadsPerMultiProcessor/32);
    return 0;
}