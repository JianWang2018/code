#include <stdio.h>
#include <hip/hip_runtime.h>
int main(int argc, char ** argv){
    hipError_t error;
    printf("%s running...\n", argv[0]);
    int   devCount;
    hipGetDeviceCount(&devCount);
    printf("number of devices: %d\n", devCount);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    printf("maxThreadsPerBlock = %d\n", devProp.maxThreadsPerBlock);
    printf("max block dimension (%d, %d, %d)\n", devProp.maxThreadsDim[0],
    devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    printf("max grid dimension (%d, %d, %d)\n", devProp.maxGridSize[0],
        devProp.maxGridSize[1], devProp.maxGridSize[2]);
    return 0;
}
