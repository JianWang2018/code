
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

struct timeval t1, t2;

#define BLOCK_SIZE 16

// kernel MM routine
__global__ void mmkernel(float *a, float *b, float  *c, int N, int M, int K)
{
  int i = threadIdx.x;
  int j = threadIdx.y;
  float sum = 0.0f;
  for (int k = 0; k< M; k++) sum += a[i+N*k] * b[k+K*j];
  c [i+N*j] = sum;
}

// host multiplication function
// C = A * B
// A is a hA x wA matrix
// B is a wA x wB matrix
// C is a hA x wB matrix

void Mul (const float *A, const float *B, float *C, int N, int M, int K)
{
  int size;
  float *dev_A, *dev_B, *dev_C;

printf("%d %d %d\n", N, M, K);

  size = N*M*sizeof(float);
  hipMalloc((void **)&dev_A, size);
  hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);

  size = M*K *sizeof(float);
  hipMalloc((void **)&dev_B, size);
  hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

  size = N*K * sizeof(float);
  hipMalloc((void **)&dev_C, size);

  dim3 dimBlock(1);
  dim3 dimGrid(N, K);
 
  mmkernel<<<dimBlock, dimGrid>>> (dev_A, dev_B, dev_C, N, M, K);

  hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);
  hipFree(dev_A);
  hipFree(dev_B);
  hipFree(dev_C);
}

int main( int argc, char *argv[])
{
  float *A, *B, *C;
  int  N, M, K, iter, i;
  int method;

  if (argc < 6) {
    printf("Usage: a.out N M K iter method\n");
    exit(0);
  }

  N= atoi(argv[1]);
  M = atoi(argv[2]);
  K = atoi(argv[3]);
  iter = 1;
  if (argc >=5) 
    iter = atoi(argv[4]);

  method = 0;
  if (argc >= 6) 
    method = atoi(argv[5]);

  A = (float *)malloc(N*M*sizeof(float));
  B = (float *)malloc(M*K*sizeof(float));
  C = (float *)malloc(N*K*sizeof(float));

  srand48(100);

  for (i=0; i<N*M; ++i) {
    //      A[i] = drand48();
    A[i] = 1.0;
    C[i] = 0.0;
  }
  for (i=0; i<M*K; ++i) {
    //      B[i] = drand48();
    //      B[i] = myid*1.0;
    B[i] = 1.0;
  }

  gettimeofday(&t1, NULL);
  for (i=0; i<iter; i++) {
    if (method == 0) {
      //      printf("A[0][0] = %lf, B[0][0] = %lf\n", *A, *B);
      Mul(A, B, C, N, M, K);
   } else {
      printf("Method not supported.\n");
      exit(0);
    }
  }
  gettimeofday(&t2, NULL);
  printf("Time for the matrix multiplication(%d) is %d milliseconds\n",
         method,
         (t2.tv_sec - t1.tv_sec)*1000 + 
         (t2.tv_usec - t1.tv_usec) / 1000);

#ifdef CHECK
  {

     FILE *fd;
     if ((fd = fopen("tmp333", "w")) == NULL) {
       printf("Cannot open tmp333\n"); exit(0);
     }
     
     for (i=0; i<N*K; i++) 
       fprintf(fd, "%6.2lf\n", C[i]);
     fclose(fd);
  }
#endif
  return 0;
}
