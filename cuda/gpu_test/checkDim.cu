#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void checkIdx() {
     int tx = threadIdx.x;
     int ty = threadIdx.y;
     int tz = threadIdx.z;
     int bx = blockIdx.x;
     int by = blockIdx.y;
     int bz = blockIdx.z;
     printf("threadIdx (%d,%d,%d), gridIdx (%d,%d,%d)\n",
             tx,ty,tz,bx,by,bz);
}
int main(){
    int  nElem = 15;
    dim3 dimBlock(4,1,1);
    dim3 dimGrid( (nElem + dimBlock.x - 1)/dimBlock.x,  1, 1);
    printf("blockdim = (%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
    printf("griddim  = (%d, %d, %d)\n", dimGrid.x,  dimGrid.y,  dimGrid.z);
    checkIdx<<<dimBlock, dimGrid>>>();
    hipDeviceReset();
    return 0;
}
